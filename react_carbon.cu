
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

/**
* Computes the log of reaction rate.
* @param a: Pointer to coefficient matrix.
* @param temp: Pointer to temperature array.
* @param lam: Matrix to write the results to.
* @param nsets: Number of sets / number of rows in coefficient matrix.
* @param ncells: Number of cells / length of temperature array.
* @param ncoeff: Number of coefficients / number of columns in coefficient matrix.
*/
template <class dtype>
__device__ void rates(dtype *a, dtype *temp, dtype *lam, int nsets, int ncells, int ncoeff)
{
    int istart = blockIdx.x * blockDim.x + threadIdx.x;
    int istep = blockDim.x * gridDim.x;
    
    int jstart = blockIdx.y * blockDim.y + threadIdx.y;
    int jstep = blockDim.y * gridDim.y;
    
    int kstart = blockIdx.z * blockDim.z + threadIdx.z;
    int kstep = blockDim.z * gridDim.z;
    
    for(int i = istart; i < nsets; i += istep)
    {
        for(int j = jstart; j < ncells; j += jstep)
        {
            dtype temp9 = temp[j] * 1e-9;
            
            for(int k = kstart; k < ncoeff; k += kstep)
            {
                switch(k)
                {
                    case 0:
                        atomicAdd(&lam[i * ncells + j], a[i * ncoeff + k]);
                        break;
                    case 6: 
                        atomicAdd(&lam[i * ncells + j], a[i * ncoeff + k] * log(temp9)); 
                        break;
                    default: 
                        atomicAdd(&lam[i * ncells + j], a[i * ncoeff + k] * pow(temp9, (2 * k - 5) / 3.0)); 
                        break;
                }
            }
        }
    }
}

template <>
__device__ void rates<float>(float *a, float *temp, float *lam, int nsets, int ncells, int ncoeff)
{
    int istart = blockIdx.x * blockDim.x + threadIdx.x;
    int istep = blockDim.x * gridDim.x;
    
    int jstart = blockIdx.y * blockDim.y + threadIdx.y;
    int jstep = blockDim.y * gridDim.y;
    
    int kstart = blockIdx.z * blockDim.z + threadIdx.z;
    int kstep = blockDim.z * gridDim.z;
    
    for(int i = istart; i < nsets; i += istep)
    {
        for(int j = jstart; j < ncells; j += jstep)
        {
            float temp9 = temp[j] * 1e-9;
            
            for(int k = kstart; k < ncoeff; k += kstep)
            {
                switch(k)
                {
                    case 0:
                        atomicAdd(&lam[i * ncells + j], a[i * ncoeff + k]);
                        break;
                    case 6:
                        atomicAdd(&lam[i * ncells + j], a[i * ncoeff + k] * logf(temp9)); 
                        break;
                    default:
                        atomicAdd(&lam[i * ncells + j], a[i * ncoeff + k] * powf(temp9, (2 * k - 5) / 3.0f)); 
                        break;
                }
            }
        }
    }
}

template <class dtype, int nsets, int ncells, int ncoeff>
__global__ void exec(dtype *lam)
{
    int xInd = blockIdx.x * blockDim.x + threadIdx.x;
    int yInd = blockIdx.y * blockDim.y + threadIdx.y;
    int ySize = blockDim.y * gridDim.y;
    int zInd = blockIdx.z * blockDim.z + threadIdx.z;
    int zSize = blockDim.z * gridDim.z;
    
    int ind = xInd * ySize * zSize + yInd * zSize + zInd;
    
    // Tensors
    __shared__ dtype a[nsets * ncoeff];
    
    // These are all of the sets in reaclib with two nuclei as reactants
    // where one of them is carbon-12.
    if(ind == 0)
    {
        // he4 + c12 -> o16 (1)
        a[0] = 69.6526;
        a[1] = -1.39254;
        a[2] = 58.9128;
        a[3] = -148.273;
        a[4] = 9.08324;
        a[5] = -0.541041;
        a[6] = 70.3554;
        
        // he4 + c12 -> o16 (2)
        a[7] = 254.634;
        a[8] = -1.84097;
        a[9] = 103.411;
        a[10] = -420.567;
        a[11] = 64.0874;
        a[12] = -12.4624;
        a[13] = 137.303;
        
        // he4 + c12 -> n + o15
        a[14] = 17.0115;
        a[15] = -98.6615;
        a[16] = 0.0;
        a[17] = 0.124787;
        a[18] = 0.0588937;
        a[19] = -0.00679206;
        a[20] = 0.0;
        
        // he4 + c12 -> p + n15 (1)
        a[21] = 27.118;
        a[22] = -57.6279;
        a[23] = -15.253;
        a[24] = 1.59318;
        a[25] = 2.4479;
        a[26] = -2.19708;
        a[27] = -0.666667;
        
        // he4 + c12 -> p + n15 (2)
        a[28] = -5.2319;
        a[29] = -59.6491;
        a[30] = 0.0;
        a[31] = 30.8497;
        a[32] = -8.50433;
        a[33] = -1.54426;
        a[34] = -1.5;
        
        // he4 + c12 -> p + n15 (3)
        a[35] = 20.5388;
        a[36] = -65.034;
        a[37] = 0.0;
        a[38] = 0.0;
        a[39] = 0.0;
        a[40] = 0.0;
        a[41] = -1.5;
        
        // he4 + c12 -> p + n15 (4)
        a[42] = -6.93365;
        a[43] = -58.7917;
        a[44] = 0.0;
        a[45] = 22.7105;
        a[46] = -2.90707;
        a[47] = 0.205754;
        a[48] = -1.5;
        
        // c12 + c12 -> n + mg23
        a[49] = -12.8056;
        a[50] = -30.1498;
        a[51] = 0.0;
        a[52] = 11.4826;
        a[53] = 1.82849;
        a[54] = -0.34844;
        a[55] = 0.0;
        
        // c12 + c12 -> p + na23
        a[56] = 60.9649;
        a[57] = 0.0;
        a[58] = -84.165;
        a[59] = -1.4191;
        a[60] = -0.114619;
        a[61] = -0.070307;
        a[62] = -0.666667;
        
        // c12 + c12 -> he4 + ne20
        a[63] = 61.2863;
        a[64] = 0.0;
        a[65] = -84.165;
        a[66] = -1.56627;
        a[67] = -0.0736084;
        a[68] = -0.072797;
        a[69] = -0.666667;
    }
    
    __shared__ dtype temp[ncells];
    
    if(ind == 0)
    {
        temp[0] = 0.5e9;
        temp[1] = 1e9;
        temp[2] = 2e9;
        temp[3] = 3e9;
    }
    
    __syncthreads();
    
    /*******************************************
    * Compute ln(lambda) for each set and cell *
    *******************************************/
    rates<dtype>(a, temp, lam, nsets, ncells, ncoeff);
}

int main()
{
    // Tensor dimensions
    const int nsets = 10, ncells = 4, ncoeff = 7;
    
    // Results matrix
    float *lam;
    hipMallocManaged(&lam, nsets * ncells * sizeof(float));
    
    for(int i = 0; i < nsets; i++)
    {
        for(int j = 0; j < ncells; j++)
        {
            lam[i * ncells + j] = 0.0f;
        }
        
        printf("\n");
    }
    
    // Compute the rates
    dim3 threadsPerBlock(nsets, ncells, ncoeff);
    dim3 numBlocks(1, 1, 1);
    exec<float, nsets, ncells, ncoeff><<<numBlocks, threadsPerBlock>>>(lam);
    
    // Print ln(lambda)
    hipDeviceSynchronize();
    printf("lambda:\n");
    
    for(int i = 0; i < nsets; i++)
    {
        for(int j = 0; j < ncells; j++)
        {
            printf("%8.3f   ", lam[i * ncells + j]);
        }
        
        printf("\n");
    }
    
    return 0;
}
